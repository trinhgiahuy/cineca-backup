#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define REPEAT 1

__global__ void arrayFunc(float* d_idata, float* d_jdata, float* d_odata, int size)
{
  int tid =  blockDim.x * blockIdx.x + threadIdx.x; 
  if (tid < size) {
    for(int i=0; i < REPEAT; i++)
       d_odata[tid] = d_idata[tid] * __expf(d_jdata[tid]);
  }
}

void initArrayData(float * array, float alpha, int size);
void arrayFuncCPU(const float* h_idata, const float* h_jdata, float* h_odata, int size);
void getChunkInfo(int i, int *d_offset, int *chunk_size, int *h_offset, int *chunk_stream, int nSize, int chunk_size_max, int num_chunk, int num_streams);

#define NSIZE 2097152
#define CHUNKSIZEMAX 65536
#define NUMSTREAMS 8

int 
main (void) {
  
  float *h_a, *h_b, *h_c;
  float *d_a, *d_b, *d_c;
  
  int nsize = NSIZE;
  int nThreads = 256;
  int nBlocks;

  hipEvent_t start, end;
  float eventEtime;

  int chunk_size_max = CHUNKSIZEMAX;
  int num_streams = NUMSTREAMS;
  int num_chunk;
  int i;
  int h_offset, d_offset;
  int chunk_size, chunk_stream;

  hipStream_t streams[NUMSTREAMS];

  // chunk number calculation
  num_chunk = (nsize-1) / chunk_size_max + 1;

  printf("Number of elements: %d\n", nsize);
  printf("Number of streams:  %d\n", num_streams);
  printf("Number of chunks:   %d\n", num_chunk);


  // allocation and initialization of host buffers
  hipHostMalloc((void**)&h_a, nsize * sizeof(float), hipHostMallocDefault);
  hipHostMalloc((void**)&h_b, nsize * sizeof(float), hipHostMallocDefault);
  hipHostMalloc((void**)&h_c, nsize * sizeof(float), hipHostMallocDefault);

  initArrayData(h_a, 1.0f, nsize);
  initArrayData(h_b, 10.0f, nsize);

//-- insert CUDA code ----------------
  // device buffers allocation

  // streams creation

//------------------------------------

  // creation of cuda events: start, end
  hipEventCreate(&start);  
  hipEventCreate(&end);

  printf ("\nGPU computation ... ");

  hipEventRecord(start,0);  

  for (i = 0; i < num_chunk; i++) {

    // please see getChunkInfo function description
    getChunkInfo(i, &d_offset, &chunk_size, &h_offset, &chunk_stream, nsize, chunk_size_max, num_chunk, num_streams);

//-- insert CUDA code ----------------
    // host to device buffer copies

//------------------------------------

    // block number calculation
    nBlocks = (chunk_size-1) / nThreads + 1;

//-- insert CUDA code ----------------
    // arrayFunc kernel launch

//------------------------------------

//-- insert CUDA code ----------------
    // copy back of results from device

//------------------------------------

  }

  hipDeviceSynchronize();

  hipEventRecord(end,0);  
  hipEventSynchronize(end);
  hipEventElapsedTime(&eventEtime, start, end);

  printf ("ok\n");

  printf("Elapsed time on GPU: %.2f ms\n", eventEtime);

  // host computation
  printf("\nCPU computation ... ");
  float *cpuResult;
  float eventTimeCPU;
  hipHostMalloc((void**)&cpuResult, nsize * sizeof(float), hipHostMallocDefault);
  hipEventRecord(start,0);

  arrayFuncCPU(h_a, h_b, cpuResult, nsize);

  hipEventRecord(end,0);
  hipEventSynchronize(end);
  hipEventElapsedTime(&eventTimeCPU, start, end);
  printf ("ok\n");
  printf("Elapsed time on CPU: %.2f ms\n", eventTimeCPU);
  printf("\nSpeed UP CPU/GPU %.1fx\n", eventTimeCPU/eventEtime);

  printf("\nCheck results:\n");
  printf ("h_c[0]       = %f\n", h_c[0]);
  printf ("cpuResult[0] = %f\n", cpuResult[0]);

  // free resources on device
  for (i = 0; i< num_streams; i++)
    hipStreamDestroy(streams[i]);
  hipEventDestroy(start);
  hipEventDestroy(end);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  // free resources on host
  hipHostFree(h_a);
  hipHostFree(h_b);
  hipHostFree(h_c);

  return 0;
}


void 
initArrayData(float * array, float alpha, int size)
{ 
  int i;
  for (i=0; i< size; i++) 
    array[i] = alpha * (float) rand() / (float) RAND_MAX;
    
}

// getChunkInfo is used to compute some useful information starting
//   from the i-th chunk, the total number of used chunks, 
//   the maximum chunk size and the array size to process
// getChunkInfo returns:
// * chunk_size: the number of elements to use in current chunk
// * chunk_stream: the stream to use to process i-th chunk
// * the X_offsets to use for accessing the correct elements of host 
//   and device arrays in data movements and kernel launch
//
void getChunkInfo(int i, int *d_offset, int *chunk_size, int *h_offset, int *chunk_stream, int nSize, int chunk_size_max, int num_chunk, int num_streams){

  int Reminder = nSize%chunk_size_max;

  *h_offset = i*chunk_size_max;
  *chunk_stream = i%num_streams;
  *chunk_size   = chunk_size_max;
  *d_offset = *chunk_stream * chunk_size_max;

  if (Reminder && (i == num_chunk-1)) *chunk_size = Reminder;

}

void arrayFuncCPU(const float* h_idata, const float* h_jdata, float* h_odata, int size)
{
  int i, j; 
  for (i=0; i<size; i++)
     for (j=0; j<REPEAT; j++)
        h_odata[i] = h_idata[i] * expf(h_jdata[i]);
}

