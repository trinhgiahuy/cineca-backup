#include "hip/hip_runtime.h"
#include <stdio.h>

#define MATRIXSIZE 448

// Each block transposes/copies a tile of TILE_DIM x TILE_DIM elements
// TILE_DIM must be an integral multiple of BLOCK_ROWS

#define TILE_DIM    32

// Number of repetitions used for timing averages. 
#define NUM_REPS  100

// OUTER: repeat over kernel launches 
// INNER: repeat inside the kernel over just the loads and stores
#define INNER yes
//#define OUTER yes

#define CUDA_CHECK( call )               \
{                                       \
hipError_t cuerror = call;              \
if ( hipSuccess != hipSuccess )            \
   printf ("CUDA ERROR in %s %s : %s\n", __FILE__, __LINE__, hipGetErrorString( cuerror ));                        \
}


// -------------------------------------------------------
// Copies
// width and height must be integral multiples of TILE_DIM
// -------------------------------------------------------

__global__ void copy(float *odata, float* idata, int width, int height, int nreps)
{
  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  
  int index  = xIndex + width*yIndex;
  for (int r=0; r < nreps; r++) {
    odata[index] = idata[index];
  }
}


// -------------------------------------------------------
// Transposes
// width and height must be integral multiples of TILE_DIM
// -------------------------------------------------------

__global__ void transposeNaive(float *odata, float* idata, int width, int height, int nreps)
{
  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

  int index_in  = xIndex + width * yIndex;
  int index_out = yIndex + height * xIndex;
  for (int r=0; r < nreps; r++) {
    odata[index_out] = idata[index_in];
  }
}

// coalesced transpose (with bank conflicts)

__global__ void transposeCoalesced(float *odata, float *idata, int width, int height, int nreps)
{
  __shared__ float tile[TILE_DIM][TILE_DIM];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;  
  int index_in = xIndex + (yIndex)*width;

  xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
  yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex + (yIndex)*height;

  for (int r=0; r < nreps; r++) {
    tile[threadIdx.y][threadIdx.x] = idata[index_in];
  
    __syncthreads();
  
    odata[index_out] = tile[threadIdx.x][threadIdx.y];
  }
}

// Coalesced transpose with no bank conflicts

__global__ void transposeNoBankConflicts(float *odata, float *idata, int width, int height, int nreps)
{
  __shared__ float tile[TILE_DIM][TILE_DIM+1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;  
  int index_in = xIndex + (yIndex)*width;

  xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
  yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex + (yIndex)*height;

  for (int r=0; r < nreps; r++) {
    tile[threadIdx.y][threadIdx.x] = idata[index_in];
  
    __syncthreads();
  
    odata[index_out] = tile[threadIdx.x][threadIdx.y];
  }
}


// ---------------------
// host utility routines
// ---------------------

void computeTransposeGold(float* gold, float* idata,
			  const  int size_x, const  int size_y)
{
  for(  int y = 0; y < size_y; ++y) {
    for(  int x = 0; x < size_x; ++x) {
      gold[(x * size_y) + y] = idata[(y * size_x) + x];
    }
  }
}

bool compare_results(float* gold, float* odata, int size)
{
  float thresold = 1e-3;
  for(  int x = 0; x < size; ++x) {
    if (abs(gold[x]-odata[x]) > thresold) {
      return false;
    }
  }
  return true;
}

int main( int argc, char** argv) 
{
  int size_x = MATRIXSIZE;
  int size_y = MATRIXSIZE;

  if (size_x%TILE_DIM != 0 || size_y%TILE_DIM != 0) {
    printf("\nMatrix size must be integral multiple of tile size\nExiting...\n\n");
    printf("FAILED\n\n");
    return 1;
  }

  // kernel pointer and descriptor
  void (*kernel)(float *, float *, int, int, int);
  char *kernelName;

  // execution configuration parameters
  dim3 grid(size_x/TILE_DIM, size_y/TILE_DIM), threads(TILE_DIM,TILE_DIM);

  // CUDA events
  hipEvent_t start, stop;

  // size of memory required to store the matrix
  const  int mem_size = sizeof(float) * size_x*size_y;

  // allocate host memory
  float *h_idata = (float*) malloc(mem_size);
  float *h_odata = (float*) malloc(mem_size);
  float *transposeGold = (float *) malloc(mem_size);  
  float *gold;

  // allocate device memory
  float *d_idata, *d_odata;
  CUDA_CHECK( hipMalloc( (void**) &d_idata, mem_size) );
  CUDA_CHECK( hipMalloc( (void**) &d_odata, mem_size) );

  // initalize host data
  for(  int i = 0; i < (size_x*size_y); ++i)
    h_idata[i] = (float) i;
  
  // copy host data to device
  CUDA_CHECK( hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice) );

  // Compute reference transpose solution
  computeTransposeGold(transposeGold, h_idata, size_x, size_y);

  // print out common data for all kernels
  printf("\nMatrix size: %dx%d (%dx%d tiles), tile size: %dx%d, block size: %dx%d\n\n", 
	 size_x, size_y, size_x/TILE_DIM, size_y/TILE_DIM, TILE_DIM, TILE_DIM, TILE_DIM, TILE_DIM);

  // initialize events
  CUDA_CHECK( hipEventCreate(&start) );
  CUDA_CHECK( hipEventCreate(&stop) );

  //
  // loop over different kernels
  //

  bool success = true;

  for (int k = 0; k<4; k++)
  {
    // set kernel pointer
    switch (k) {
    case 0:
      kernel = &copy;                           kernelName = "simple copy       "; break;
    case 1:
      kernel = &transposeNaive;                 kernelName = "naive             "; break;
    case 2:
      kernel = &transposeCoalesced;             kernelName = "coalesced         "; break;
    case 3:
      kernel = &transposeNoBankConflicts;       kernelName = "optimized         "; break;
    }      

    // set reference solution
    if (kernel == &copy) {
      gold = h_idata;
    } else {
      gold = transposeGold;
    }

    // warmup to avoid timing startup
    kernel<<<grid, threads>>>(d_odata, d_idata, size_x, size_y, 1);
    bool res;
#ifdef OUTER
    // take measurements for loop over kernel launches
    CUDA_CHECK( hipEventRecord(start, 0) );
    for (int i=0; i < NUM_REPS; i++) {
      kernel<<<grid, threads>>>(d_odata, d_idata, size_x, size_y, 1);
    }
    CUDA_CHECK( hipEventRecord(stop, 0) );
    CUDA_CHECK( hipEventSynchronize(stop) );
    float outerTime;
    CUDA_CHECK( hipEventElapsedTime(&outerTime, start, stop) );    

    CUDA_CHECK( hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost) );
    res = compare_results(gold, h_odata, size_x*size_y);
    if (res == false) {
      printf("*** %s kernel FAILED ***\n", kernelName);
      success = false;
    }

    // report effective bandwidths
    float outerBandwidth = 2.0f * 1000.0f * mem_size/(1024*1024*1024)/(outerTime/NUM_REPS);

    printf("transpose-Outer-%s, Throughput = %9.4f GB/s, Time = %.5f s, Size = %u fp32 elements\n", 
           kernelName, outerBandwidth, outerTime/NUM_REPS, (size_x * size_y));
#endif

#ifdef INNER
    // take measurements for loop inside kernel
    CUDA_CHECK( hipEventRecord(start, 0) );
    kernel<<<grid, threads>>>(d_odata, d_idata, size_x, size_y, NUM_REPS);
    CUDA_CHECK( hipEventRecord(stop, 0) );
    CUDA_CHECK( hipEventSynchronize(stop) );
    float innerTime;
    CUDA_CHECK( hipEventElapsedTime(&innerTime, start, stop) );    

    CUDA_CHECK( hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost) );
    res = compare_results(gold, h_odata, size_x*size_y);
    if (res == false) {
      printf("*** %s kernel FAILED ***\n", kernelName);
      success = false;
    }
    
    // report effective bandwidths
    float innerBandwidth = 2.0f * 1000.0f * mem_size/(1024*1024*1024)/(innerTime/NUM_REPS);

    printf("transpose-Inner-%s, Throughput = %9.4f GB/s, Time = %.5f s, Size = %u fp32 elements\n", 
           kernelName, innerBandwidth, innerTime/NUM_REPS, (size_x * size_y));
#endif
   }
  
  printf("\n%s\n\n", (success == true) ? "PASSED" : "FAILED");

  // cleanup
  free(h_idata);
  free(h_odata);
  free(transposeGold);
  CUDA_CHECK( hipFree(d_idata) );
  CUDA_CHECK( hipFree(d_odata) );

  CUDA_CHECK( hipEventDestroy(start) );
  CUDA_CHECK( hipEventDestroy(stop) );
  
  hipDeviceReset();
  return 0;

}
