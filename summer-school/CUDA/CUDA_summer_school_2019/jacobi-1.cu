
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

// Grid boundary conditions
#define RIGHT 1.0
#define LEFT 1.0
#define TOP 1.0
#define BOTTOM 10.0

// Algorithm settings
#define NPRINT 1000
#define MAX_ITER 100000
#define TOLERANCE 0.0001

void grid_init(double *grid,double *grid_new,int nx, int ny); 

// CUDA kernel
__global__
void stencil_sum(double*grid, double *grid_new, int nx, int ny)
{
  int index=blockIdx.x * blockDim.x +threadIdx.x; // global thread id

  int nrow=index/ny;
  int diff=index-(nrow*ny);
  int k=(nrow+1)*(ny+2)+diff+1;

  if (index<nx*ny)
      grid_new[k]=0.25 * (grid[k-1]+grid[k+1] + grid[k-(ny+2)] + grid[k+(ny+2)]);
}

// kernel for performing the normalisation

__global__
void stencil_norm(double*grid, double*arraynorm, int nx, int ny)
{
  int index=blockIdx.x * blockDim.x +threadIdx.x; // globEl thread id

  int nrow=index/ny;
  int diff=index-(nrow*ny);
  int k=(nrow+1)*(ny+2)+diff+1;

  if (index<nx*ny)
     arraynorm[index]=(double)pow(grid[k]*4.0-grid[k-1]-grid[k+1] - grid[k-(ny+2)] - grid[k+(ny+2)], 2);

}


int main(int argc, char*argv[]) {

  int i,j,k;
  double tmpnorm,bnorm,norm;

  if (argc !=3) {
      printf("usage: $argv[0] GRIDX GRIDY \n");
      return(1);
  }

  int nx=atoi(argv[1]);
  int ny=atoi(argv[2]);
  int nelems=(nx+2)*(ny+2);

  printf("grid size %d X %d \n",ny,ny);

// Allocate memory for current and new grids
  double *grid= (double*)malloc(sizeof(double)*nelems);
  double *grid_new= (double*)malloc(sizeof(double)*nelems);
  double *arraynorm= (double*)malloc(sizeof(double)*nx*ny);

  // initialise grids
  grid_init(grid,grid_new,nx,ny); 

  // calculate norm factor
  tmpnorm=0.0;
  for (i=1;i<=nx;i++) {
    for (j=1;j<=ny;j++) {
      k=(ny+2)*i+j;            
      tmpnorm=tmpnorm+(double)pow(grid[k]*4.0-grid[k-1]-grid[k+1] - grid[k-(ny+2)] - grid[k+(ny+2)], 2); 
    }
  }
  bnorm=sqrt(tmpnorm);

//  CUDA
//  Allocate device memory.
  double *d_grid, *d_grid_new,*d_arraynorm;
  hipMalloc(&d_grid,nelems*sizeof(double));
  hipMalloc(&d_grid_new,nelems*sizeof(double));
  hipMalloc(&d_arraynorm,nx*ny*sizeof(double));

  hipMemcpy(d_grid_new,grid_new,nelems*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_grid,grid,nelems*sizeof(double), hipMemcpyHostToDevice);


  // set kernel parameters
  int blockSize=256;
  int numBlocks = ((nx*ny)+blockSize-1)/blockSize;
  printf("numBlocks=%d\n",numBlocks);


//    MAIN LOOP 
  int iter;
  for (iter=0; iter<MAX_ITER; iter++) {


   stencil_norm<<<numBlocks,blockSize>>>(d_grid,d_arraynorm,nx,ny);
   hipMemcpy(arraynorm,d_arraynorm,nx*ny*sizeof(double),hipMemcpyDeviceToHost);   
  
   tmpnorm=0.0;
   for (i=0;i<nx*ny;i++)
      tmpnorm=tmpnorm+arraynorm[i];
 
    // calculate norm factor
    norm=(double)sqrt(tmpnorm)/bnorm;

    if (norm < TOLERANCE) break;

    // CUDA
    // grid update sent to GPU

    stencil_sum<<<numBlocks,blockSize>>>(d_grid,d_grid_new,nx,ny);

  // Wait for GPU to finish
   hipDeviceSynchronize();

    double *temp=d_grid_new;
    d_grid_new=d_grid;
    d_grid=temp;

    if (iter % NPRINT ==0) printf("Iteration =%d ,Relative norm=%e\n",iter,norm);
  }

  printf("Terminated on %d iterations, Relative Norm=%e \n", iter,norm);
  
  // free memory resources
  free(grid);
  free(grid_new);
  free(arraynorm);

  //CUDA
  // Free CUDA resources 
  hipFree(d_grid);
  hipFree(d_grid_new);
  hipFree(d_arraynorm);


  return 0;
    

  }

// Initialise Grids 
void grid_init(double *grid,double *grid_new,int nx, int ny) {

  int i,j,k;

  // top and bottom boundaries
  for (i=0;i<ny+2;i++) {
    grid_new[i]=grid[i]=TOP;
    j=(ny+2)*(nx+1)+i;
    grid_new[j]=grid[j]=BOTTOM;
  }

  // left and right boundaries
  for (i=1;i<nx+1;i++) {
    j=(ny+2)*i;
    grid_new[j]=grid[j]=LEFT;
    grid_new[j+ny+1]=grid[j+ny+1]=RIGHT;
  }

  // Initialise rest of grid
  for (i=1;i<=nx;i++)
    for (j=1;j<=ny;j++) {
      k=(ny+2)*i+j;
      grid_new[k]=grid[k]=0.0;
    }

} // end grid_init


