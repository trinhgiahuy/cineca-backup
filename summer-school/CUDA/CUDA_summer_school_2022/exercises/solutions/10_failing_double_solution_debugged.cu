
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N) {
  int i;
  for (i = 0; i < N; ++i)
    a[i] = i;
}

__global__
void doubleElements(int *a, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for (int i = idx; i < N; i += stride)
    a[i] *= 2;
}

bool checkElementsAreDoubled(int *a, int N) {
  int i;
  for (i = 0; i < N; ++i) {
    if (a[i] != i*2)
      return false;
  }
  return true;
}

int main()
{
  int N = 10000;
  int *a;

  size_t size = N * sizeof(int);
  hipMallocManaged(&a, size);

  init(a, N);

  size_t threads_per_block = 1024;
  size_t number_of_blocks = 32;

  hipError_t syncErr, asyncErr;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);

  syncErr = hipGetLastError();
  asyncErr = hipDeviceSynchronize();

  if (syncErr != hipSuccess) printf("ErrorSync: %s\n", hipGetErrorString(syncErr));
  if (asyncErr != hipSuccess) printf("ErrorAsync: %s\n", hipGetErrorString(asyncErr));

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}
  