
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void printSuccessForCorrectExecutionConfiguration() {
  if (threadIdx.x == 1023 && blockIdx.x == 255) {
        printf("Success!\n");
  }
}

int main()
{
  printSuccessForCorrectExecutionConfiguration<<<1, 1>>>();
  hipDeviceSynchronize();
}
